#include "TimerGPU.h"


CTimerGPU::CTimerGPU(void)
{
	hipEventCreate(&m_start);
	hipEventCreate(&m_current);
}


CTimerGPU::~CTimerGPU(void)
{
	hipEventDestroy(m_start);
	hipEventDestroy(m_current);
}


void CTimerGPU::Start(void)
{
	m_bStarted = true;

	hipEventRecord(m_start, 0);
}


float CTimerGPU::GetElapsedTimeInMilliseconds(void)
{
	if (m_bStarted)
	{
		hipEventRecord(m_current, 0);
		hipEventSynchronize(m_current);

		float fElapsedTime;
		hipEventElapsedTime(&fElapsedTime, m_start, m_current);

		return fElapsedTime;
	}

	return 0.0f;
}
